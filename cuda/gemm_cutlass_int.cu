#include <cutlass/numeric_types.h>
#include <cutlass/gemm/device/gemm.h>

#include <cutlass/util/host_tensor.h>


#include <iostream>
#include <gemm.cuh>

#include <cstdint>


template <typename Type>
void gemm_cutlass(int M, int N, int K, Type *X, Type *W, Type *Y) {
    Type *x, *w, *y;
    hipMalloc((void **)&x, M * K * sizeof(Type)); //Matrix X has M rows and K columns
    hipMalloc((void **)&w, K * N * sizeof(Type)); //Matrix W has K rows and N columns
    hipMalloc((void **)&y, M * N * sizeof(Type)); //Matrix Y has M rows and N columns

    hipMemcpy(x, X, M * K * sizeof(Type), hipMemcpyHostToDevice);
    hipMemcpy(w, W, K * N * sizeof(Type), hipMemcpyHostToDevice);
    hipMemcpy(y, Y, M * N * sizeof(Type), hipMemcpyHostToDevice);

    gpu::gemm<Type>(M, N, K, x, true, w, true, y, true); //true means that the matrix is in row-major order

    hipMemcpy(Y, y, M * N * sizeof(Type), hipMemcpyDeviceToHost);

    hipFree(x);
    hipFree(w);
    hipFree(y);
}

//forward declare the function for different integer Testtypes
template void gemm_cutlass<int8_t>(int M, int N, int K, int8_t *X, int8_t *W, int8_t *Y);
template void gemm_cutlass<int16_t>(int M, int N, int K, int16_t *X, int16_t *W, int16_t *Y);
template void gemm_cutlass<int32_t>(int M, int N, int K, int32_t *X, int32_t *W, int32_t *Y);
template void gemm_cutlass<int64_t>(int M, int N, int K, int64_t *X, int64_t *W, int64_t *Y);
template void gemm_cutlass<unsigned int>(int M, int N, int K, unsigned int *X, unsigned int *W, unsigned int *Y);
